#include "hip/hip_runtime.h"
#include "cudaSegEngine.h"
#include "cudaUtil.h"
#include "cudaImgTrans.h"
#include "cudaSegSLIC.h"

#include <stdio.h>
#include <math.h>
#include <time.h>

using namespace std;

__device__ uchar4*      rgbBuffer;
__device__ float4*      floatBuffer;
__device__ int32_t*     maskBuffer;

int nWidth,nHeight,nSeg,nMaxSegs;
bool cudaIsInitialized=false;

__device__ SLICClusterCenter* vSLICCenterList;
bool slicIsInitialized=false;

__host__ void InitCUDA(int width, int height,int nSegment, SEGMETHOD eMethod)
{
	//for all methods
	if (!cudaIsInitialized)
	{
		nWidth=width;
		nHeight=height;

		hipMalloc((void**) &rgbBuffer,width*height*sizeof(uchar4));
		hipMalloc((void**) &floatBuffer,width*height*sizeof(float4));
        hipMalloc((void**) &maskBuffer,width*height*sizeof(int32_t));

		hipMemset(floatBuffer,0,width*height*sizeof(float4));
        hipMemset(maskBuffer,0,width*height*sizeof(int32_t));

		nSeg=nSegment;
		cudaIsInitialized=true;
	}

	if (!slicIsInitialized)
	{
		// MaxSegs should be same on initializeFastSeg()@FastImgSeg.cpp
		nMaxSegs=(iDivUp(nWidth,BLCK_SIZE)*2)*(iDivUp(nHeight,BLCK_SIZE)*2);

		// the actual number of segments
		hipMalloc((void**) &vSLICCenterList,nMaxSegs*sizeof(SLICClusterCenter));
		hipMemset(vSLICCenterList,0,nMaxSegs*sizeof(SLICClusterCenter));
		slicIsInitialized=true;
	}
}

extern "C" __host__ void CUDALoadImg(unsigned char* imgPixels)
{
	if (cudaIsInitialized)
	{
		hipMemcpy(rgbBuffer,imgPixels,nWidth*nHeight*sizeof(uchar4),hipMemcpyHostToDevice);
	}
	else
	{
		return;
	}
}

__host__ void TerminateCUDA()
{
	if (cudaIsInitialized)
	{
		hipFree(rgbBuffer);
		hipFree(floatBuffer);
		hipFree(maskBuffer);

		cudaIsInitialized=false;
	}

	if (slicIsInitialized)
	{
		hipFree(vSLICCenterList);

		slicIsInitialized=false;
	}
}

__host__ void CudaSegmentation(SEGMETHOD eSegmethod, double weight)
{
	switch (eSegmethod)
	{
	case SLIC :

		Rgb2CIELab(rgbBuffer,floatBuffer,nWidth,nHeight);
		SLICImgSeg(maskBuffer,floatBuffer,nWidth,nHeight,nSeg,vSLICCenterList,nMaxSegs,(float)weight);

		break;

	case RGB_SLIC:

		Uchar4ToFloat4(rgbBuffer,floatBuffer,nWidth,nHeight);
		SLICImgSeg(maskBuffer,floatBuffer,nWidth,nHeight,nSeg,vSLICCenterList,nMaxSegs,(float)weight);

		break;

	case XYZ_SLIC:

		Rgb2XYZ(rgbBuffer,floatBuffer,nWidth,nHeight);
		SLICImgSeg(maskBuffer,floatBuffer,nWidth,nHeight,nSeg,vSLICCenterList,nMaxSegs,(float)weight);

		break;
	}

	hipDeviceSynchronize();
}

__host__ void CopyImgDeviceToHost( unsigned char* imgPixels, int width, int height)
{
	if (cudaIsInitialized)
	{
		hipMemcpy(imgPixels,rgbBuffer,nHeight*nWidth*sizeof(uchar4),hipMemcpyDeviceToHost);
	}
}

__host__ void CopyMaskDeviceToHost( int32_t* maskPixels)
{
	if (cudaIsInitialized)
	{
        hipMemcpy(maskPixels,maskBuffer,nHeight*nWidth*sizeof(int32_t),hipMemcpyDeviceToHost);
	}
}

__host__ void CopyCenterListDeviceToHost(SLICClusterCenter* centerList)
{
	if (cudaIsInitialized)
	{
		hipMemcpy(centerList,vSLICCenterList,nMaxSegs*sizeof(SLICClusterCenter),hipMemcpyDeviceToHost);
	}
}
